
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include <cstdlib>
#include <string>
#include <sstream>
#include <iostream>
#include <fstream>
#include <vector>
#include <cstring>

/**
 * @brief Used to make it easier to pass the known data between functions.
 * 
 *  Knowledge.positions is a list of 5 characters, which are only initialized once the character at that 
 * position of the solution is known
 * 
 * Knowledge.letter_counts is a list of 26 shorts which represent the coutns of each letter of the alphabet that are
 * known. The count for a letter is set to -1 if it is known that that letter is not in the solution
 */
struct Knowledge {
    char state[5];
    short letter_counts[26];
};

/**
 * @brief Stores data about a particular call of the solve() function
 */
struct GuessResults {
    std::string solution;
    float seconds;
    bool solved;
    int num_guesses;
};

/**
 * @brief Randomizes a list of words
 * 
 * @param list std::vector<std::string>, a list of words
 * @return std::vector<std::string>, the list, randomized
 */
std::vector<std::string> randomize_list(std::vector<std::string> list) {
    std::vector<std::string> randomized = {};

    while (list.size() > 0) {
        int i = std::rand() % list.size();
        randomized.push_back(list[i]);
        list.erase(list.begin() + i);
    }

    return randomized;
}

/**
 * @brief Opens a list of words stored in a file (line-separated) and returns it as an array
 * 
 * @param path std::string, the path to the file to open
 * @param count int, the number of words to extract from the file (256 or the number of items in the file by default)
 * @return std::string*, the list of words extracted from the file
 */
std::vector<std::string> get_word_list(std::string path, int count) {
    std::vector<std::string> output;
    output.reserve(count);

    std::ifstream file(path);
    std::string word;
    while (getline(file, word) && count > 0) {
        output.push_back(word);
        count--;
    }

    return output;
}

void write_results(std::string path, GuessResults results) {
    std::ofstream file;
    file.open(path, std::ios::app);
    file << results.solution << ","
         << results.solved << ","
         << results.num_guesses << ","
         << results.seconds << "\n";
    file.close();
}

/**
 * @brief Prints a guess using the appropriate colors based on what information is known. A letter appears yellow if it appears in
 * the word but not at that position and green if it is at that position, otherwie gray if the letter is not in the word
 * 
 * @param known Knowledge, the knowledge known at the point the guess was made
 * @param guess std::string, the guess to print
 */
void print_guess(Knowledge known, std::string guess) {
    for (int i = 0; i<guess.size(); i++) {
        if (guess.at(i) == known.state[i]) {
            known.letter_counts[int(guess.at(i))-97]--;
        }
    }
    
    for (int i = 0; i<guess.size(); i++) {
        if (guess.at(i) == known.state[i]) {
            std::cout << "\x1B[32m" << guess.at(i) << "\033[0m"; // ANSI Green
        } else if (known.letter_counts[int(guess.at(i))-97] > 0) {
            known.letter_counts[int(guess.at(i))-97]--;
            std::cout << "\x1B[33m" << guess.at(i) << "\033[0m"; // ANSI Yellow
        } else {
            std::cout << guess.at(i);
        }
        std::cout << " ";
    }
}

/**
 * @brief Prints a distribution graph for how well the bot did solving the game
 * 
 * @param dist 
 */
void print_dist(std::vector<int> dist) {
    std::cout << "1: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 1)
            std::cout << "#";
    std::cout << "\n2: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 2)
            std::cout << "#";
    std::cout << "\n3: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 3)
            std::cout << "#";
    std::cout << "\n4: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 4)
            std::cout << "#";
    std::cout << "\n5: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 5)
            std::cout << "#";
    std::cout << "\n6: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 6)
            std::cout << "#";
    std::cout << "\nF: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == -1)
            std::cout << "#";
    std::cout << std::endl; 
}

/**
 * @brief Extracts the information from a given word and stores it in a Knowledge struct
 * 
 * @param word std::string
 * @return Knowledge 
 */
Knowledge get_info(std::string word) {
    Knowledge known = {};
    for (int i = 0; i<5; i++) {
        known.state[i] = word.at(i);
        known.letter_counts[int(word.at(i))-97]++;
    }

    return known;
}

/**
 * @brief Updates the given Knowledge struct with new information gained from a new guess. This method assumes that the 
 * solution is known to the user and the bot is running automatically. Otherwise, the Knowledge needs to be updated manually.
 * 
 * @param known Knowledge&, the information to update
 * @param guess std::string, the new guess to get more information from
 * @param solution std::string, the solution being used at the moment
 */
void learn(Knowledge& known, std::string guess, std::string solution) {
    Knowledge s_info = get_info(solution);
    Knowledge g_info = get_info(guess);

    for (int i = 0; i<5; i++)
        if (g_info.state[i] == s_info.state[i])
            known.state[i] = g_info.state[i];

    for (int i = 0; i < 26; i++) {
        if (known.letter_counts[i]<s_info.letter_counts[i] && known.letter_counts[i]<g_info.letter_counts[i])
            known.letter_counts[i] = (g_info.letter_counts[i]<s_info.letter_counts[i])?
                                     (g_info.letter_counts[i]):(s_info.letter_counts[i]);
        if (s_info.letter_counts[i] == 0 && g_info.letter_counts[i] > 0)
            known.letter_counts[i] = -1;
    }
        
}

/**
 * @brief Culls the word list to only contain words that are still possible solutions, based on the known information
 * 
 * @param word_list std::vector<std::string>, the list to cull
 * @param known Knowledge, the known information
 */
void cull_word_list(std::vector<std::string>& word_list, Knowledge known) {
    for (int i = word_list.size()-1; i>=0; i--) {
        bool is_valid = true;
        for (int j = 0; j<5; j++) {
            if (known.letter_counts[int(word_list.at(i).at(j))-97] < 0) {
                is_valid = false;
                break;
            } else if (known.state[j] != 0 && word_list.at(i).at(j) != known.state[j]) {
                is_valid = false;
                break;
            }
        }

        for (int j = 0; j<26; j++) {
            if (known.letter_counts[j] > 0) {
                bool contains_letter = false;
                for (int k = 0; k<5; k++) {
                    if (word_list.at(i).at(k) == char(j+97)) {
                        contains_letter = true;
                    }
                }
                if (!contains_letter) {
                    is_valid = false;
                    break;
                }
            }
        }

        if (!is_valid) {
            word_list.erase(word_list.begin() + i);
        }
    }
}

/**
 * @brief Kernel function to get the letter counts from a given word return it as an array of shorts. 
 * 
 * @param word char[5], the word to get info from
 * @return short[26], the array of letter counts
 */
__device__ void d_get_letter_counts(char* word, short* letter_counts) {
    for (int i = 0; i<5; i++) {
        letter_counts[int(word[i])-97]++;
    }
}

/**
 * @brief Kernel function to get a specific 5-letter word from a flattened list of words
 * 
 * @param word_list char*, the flattened word list
 * @param idx int, the index of the word to grab
 * @return char[5] 
 */
__device__ void d_get_word(char* word_list, int idx, char* word) {
    for (int i = 0; i<5; i++) 
        word[i] = word_list[idx*5+i];
}

/**
 * @brief Kernel function to update the given state and letter counts with information gained from a new guess.
 * 
 * Comparable to the host function learn()
 * 
 * @param guess char[5], the guess
 * @param g_letters short[26], the count of letters in the guess
 * @param solution char[5], the solution
 * @param s_letters short[26], the count of letters in the solution
 * @param learned_state &char[5], the known state, updates with new information
 * @param learned_letters &short[26], the known solution letter counts, updates with new information
 */
__device__ void d_learn(char* guess, short* g_letters, char* solution, short* s_letters, 
                        char* learned_state, short* learned_letters) {
    for (int i = 0; i<5; i++)
        if (guess[i] == solution[i])
            learned_state[i] = guess[i];

    for (int i = 0; i < 26; i++) {
        if (learned_letters[i]<s_letters[i] && learned_letters[i]<g_letters[i])
            learned_letters[i] = (g_letters[i]<s_letters[i])?(g_letters[i]):(s_letters[i]);
        if (s_letters[i] == 0 && g_letters[i] > 0)
            learned_letters[i] = -1;
    }
}

/**
 * @brief Kernel function to determine how many potential solutions are excluded by the known information
 * 
 * @param word_list char*, the flattened solution list
 * @param n int, the size of the word list
 * @param known_state char[5], the known state of the solution
 * @param known_letter_counts short[26], the known letter counts in the solution
 * @return int, the number of valid guesses
 */
__device__ int d_count_exclusions(char *word_list, int n, char* known_state, short* known_letter_counts) {
    int excluded = 0;
    for (int sol_idx = 0; sol_idx<n; sol_idx++) {
        char possible_solution[5]; d_get_word(word_list, sol_idx*5, possible_solution);
        short ps_letter_counts[26]; d_get_letter_counts(possible_solution, ps_letter_counts);
        bool is_valid = true;

        // Check the state
        for (int i = 0; i<5; i++) {
            if (known_state[i] != 0 && possible_solution[i] != known_state[i]) {
                is_valid = false;
                break;
            }
        }

        // Check the letter counts if the potential solution is still valid
        if (is_valid) {
            for (int i = 0; i<26; i++) {
                // There is a letter present in the potential solution that we know is not in the actual solution
                if(known_letter_counts[i] < 0 && ps_letter_counts[i] > 0) {
                    is_valid = false;
                    break;
                } 
                // There are letters we known are in the actual solution that are not in the potential solution
                else if (ps_letter_counts[i] < known_letter_counts[i]) {
                    is_valid = false;
                    break;
                }
            }
        }

        if (!is_valid)
            excluded++;
    }

    return excluded;
}

/**
 * @brief Kernel function to get the expected information for each word in the word_list
 * 
 * @param word_list char**, the list of words to get expected information for
 * @param solution_list char**, the list of potential solutions to check each element of word_list against
 * @param n int, the number of words in the word list
 * @param k int, the number of words in the solution list
 * @param info float*, the list of expected information values (generated by this function)
 */
__global__ void get_expected_information(char *word_list, char *solution_list, int *n, int *k, float *expected_info) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < *n) {
        char guess[5]; d_get_word(word_list, idx, guess); //printf("Thread %d got the guess\n", idx);
        short g_letter_counts[26]; d_get_letter_counts(guess, g_letter_counts); //printf("Thread %d got the letter counts\n", idx);
        
        int sum_exclusions = 0;
        // Loops through each potential solution to see how many guesses from word_list would be removed
        // if it were the actual solution
        for (int sol_idx = 0; sol_idx<*k; sol_idx++) {
            char potential_solution[5]; d_get_word(solution_list, sol_idx, potential_solution);
            short ps_letter_counts[26]; d_get_letter_counts(potential_solution, ps_letter_counts);

            char state[5] = {};
            short letter_counts[26] = {};
            d_learn(guess, g_letter_counts, potential_solution, ps_letter_counts, state, letter_counts); //printf("Thread %d learned\n", idx);

            sum_exclusions += d_count_exclusions(solution_list, *n, state, letter_counts); //printf("Thread %d counted exclusions\n", idx);
        }

        expected_info[idx] = float(sum_exclusions) / float(*k);
    }
}

/**
 * @brief Makes a guess for the solution based on the amount of information that can be expected to be found by making the guess.
 * The expected information value for each 
 * 
 * @param word_list std::vector<std::string>, the list of words to guess from
 * @return std::string 
 */
std::string make_informed_guess(std::vector<std::string> word_list) {
    int size = word_list.size();
    int *n = &size;

    // Allocate and initialize host memory
    float *info = (float*)malloc(size*sizeof(float));
    char *words = (char*)malloc(size*5*sizeof(char));
    for (int i = 0; i<size*5; i++) {
        words[i] = word_list[int(i/5)].at(i%5);
    }

    // Allocate device memory
    float *d_info;
    char *d_words, *d_sols;
    int *d_n, *d_k;
    hipMalloc(&d_info, size*sizeof(float));
    hipMalloc(&d_words, size*5*sizeof(char));
    hipMalloc(&d_sols, size*5*sizeof(char));
    hipMalloc(&d_n, sizeof(int));
    hipMalloc(&d_k, sizeof(int));

    // Copy from host to device
    hipMemcpy(d_words, words, size*5*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_sols, words, size*5*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_n, n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_k, n, sizeof(int), hipMemcpyHostToDevice);

    // Kernel call
    get_expected_information<<<32, 512>>>(d_words, d_sols, d_n, d_k, d_info);
    hipDeviceSynchronize();

    // Copy data back to host
    hipMemcpy(info, d_info, size*sizeof(float), hipMemcpyDeviceToHost);

    // Interpret data
    int max_idx = 0;
    for (int i = 0; i<size; i++) 
        if (info[i] > info[max_idx]) 
            max_idx = i;

    // Free memory
    free(info); 
    free(words); 
    hipFree(d_info); 
    hipFree(d_words); 
    hipFree(d_n); 
    hipFree(d_k); 
    
    return word_list[max_idx];
}

/**
 * @brief Makes a guess for the solution at random
 * 
 * @param word_list std::vector<std::string>, the list of words to guess from
 * @return std::string, The randomly-made guess
 */
std::string make_random_guess(std::vector<std::string> word_list) {
    return word_list[std::rand() % word_list.size()];
}

/**
 * @brief Solves a wordle puzzle for a given solution
 * 
 * @param word std::string, the solution to solve for
 * @param path std::string, the file path for a list of words to use as the possible guesses
 * @return int, the number of guesses it took to solve, or -1 if it failed
 */
void solve(std::string word, std::string path, int n, GuessResults &results) {
    float total_time = 0.0;
    bool solved = false;
    short attempts = 0;
    Knowledge known = {};
    std::vector<std::string> words = randomize_list(get_word_list(path, n));

    std::cout << "Guessing '" << word << "' with expected information..." << std::endl;
    while (attempts < 6 && !solved) {
        int num_remaining = words.size();

        auto start = std::chrono::high_resolution_clock::now();
        std::string guess = make_informed_guess(words);
        auto stop = std::chrono::high_resolution_clock::now();
        auto dur = std::chrono::duration_cast<std::chrono::milliseconds>(stop-start);
        total_time += float(dur.count())/float(1000);

        learn(known, guess, word);
        int guess_idx = 0;
        for (int i = 0; i<words.size(); i++)
            if (words[i] == guess) {
                guess_idx = i;
                break;
            }
        words.erase(words.begin() + guess_idx);
        cull_word_list(words, known);

        std::cout << "     "; 
        print_guess(known, guess); 
        std::cout << "   (out of " << num_remaining << " in " << float(dur.count())/float(1000) << " seconds)\n";

        attempts++;
        if (guess == word)
            solved = true;
    }
        
    std::string message = (solved)?("Solved!"):("Failed!");
    std::cout << message << std::endl;

    results.num_guesses = attempts;
    results.solved = solved;
    results.seconds = total_time;

}

int main(int argc, char **argv) {
    if (argc != 4 && argc != 5) {
        std::cout << "Incorrect Usage: ./solver [solution] [word list path] [size] <output file path>";
        printf("\n");
        return 0;
    } 
    std::string solution = argv[1];
    std::string path = argv[2];
    int num = atoi(argv[3]);
    std::string output = ""; if (argc>3) output = argv[4];
    
    std::srand(time(0));
    printf("\n");

    GuessResults results = {}; results.solution = solution;
    solve(solution, path, num, results);
    if (output != "") 
        write_results(output, results);

    printf("\n");

    return 0;
}