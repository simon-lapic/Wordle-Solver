
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <cstdlib>
#include <string>
#include <sstream>
#include <iostream>
#include <fstream>
#include <vector>
#include <cstring>

/**
 * @brief Used to make it easier to pass the known data between functions.
 * 
 *  Knowledge.positions is a list of 5 characters, which are only initialized once the character at that 
 * position of the solution is known
 * 
 * Knowledge.letter_counts is a list of 26 shorts which represent the coutns of each letter of the alphabet that are
 * known. The count for a letter is set to -1 if it is known that that letter is not in the solution
 */
struct Knowledge {
    char state[5];
    short letter_counts[26];
};

/**
 * @brief Opens a list of words stored in a file (line-separated) and returns it as an array
 * 
 * @param path std::string, the path to the file to open
 * @param count int, the number of words to extract from the file (256 or the number of items in the file by default)
 * @return std::string*, the list of words extracted from the file
 */
std::vector<std::string> get_word_list(std::string path, int count=256) {
    std::vector<std::string> output;
    output.reserve(count);

    std::ifstream file(path);
    std::string word;
    while (getline(file, word) && count > 0) {
        output.push_back(word);
        count--;
    }

    return output;
}

/**
 * @brief Determines whether or not a string is a valid word for the wordle. A word is considered valid if it has five
 * characters, and each character is a letter.
 * 
 * @param word std::string, the word to validate
 * @return true if word is valid, otherwise false
 */
bool validate(std::string word) {
    bool valid = word.size() == 5;
    for (char c : word)
        if (!(int(c) >= 97 && int(c) <= 122)) //Confirms that the character is one of the ascii lowercase letters
            valid = false;
    return valid;
}

/**
 * @brief Prints a guess using the appropriate colors based on what information is known. A letter appears yellow if it appears in
 * the word but not at that position and green if it is at that position, otherwie gray if the letter is not in the word
 * 
 * @param known Knowledge, the knowledge known at the point the guess was made
 * @param guess std::string, the guess to print
 */
void print_guess(Knowledge known, std::string guess) {
    for (int i = 0; i<guess.size(); i++) {
        if (guess.at(i) == known.state[i]) {
            known.letter_counts[int(guess.at(i))-97]--;
        }
    }
    
    for (int i = 0; i<guess.size(); i++) {
        if (guess.at(i) == known.state[i]) {
            std::cout << "\x1B[32m" << guess.at(i) << "\033[0m"; // ANSI Green
        } else if (known.letter_counts[int(guess.at(i))-97] > 0) {
            known.letter_counts[int(guess.at(i))-97]--;
            std::cout << "\x1B[33m" << guess.at(i) << "\033[0m"; // ANSI Yellow
        } else {
            std::cout << guess.at(i);
        }
        std::cout << " ";
    }

    std::cout << std::endl;
}

/**
 * @brief Prints a distribution graph for how well the bot did solving the game
 * 
 * @param dist 
 */
void print_dist(std::vector<int> dist) {
    std::cout << "1: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 1)
            std::cout << "#";
    std::cout << "\n2: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 2)
            std::cout << "#";
    std::cout << "\n3: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 3)
            std::cout << "#";
    std::cout << "\n4: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 4)
            std::cout << "#";
    std::cout << "\n5: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 5)
            std::cout << "#";
    std::cout << "\n6: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == 6)
            std::cout << "#";
    std::cout << "\nF: ";
    for (int i = 0; i<dist.size(); i++)
        if (dist[i] == -1)
            std::cout << "#";
    std::cout << std::endl; 
}

/**
 * @brief Updates the given Knowledge struct with new information gained from a new guess. This method assumes that the 
 * solution is known to the user and the bot is running automatically. Otherwise, the Knowledge needs to be updated manually
 * 
 * @param known Knowledge&, the information to update
 * @param guess std::string, the new guess to get more information from
 * @param solution std::string, the solution being used at the moment
 */
void update_knowledge(Knowledge& known, std::string guess, std::string solution) {
    for (int i = 0; i<5; i++) {
        int count = 0;
        bool found = false;
        for (int j = 0; j<5; j++) {
            if (guess.at(i) == solution.at(j)) { // Finding the counts gotten from the guess
                count++;
                found = true;
                if (i == j) { // Update the state where necessary
                    known.state[i] = guess.at(i);
                }
            }
        }

        // Update the letter counts if necessary
        if (known.letter_counts[int(guess.at(i))-97] < count) {
            known.letter_counts[int(guess.at(i))-97] = count;
        }

        if (!found) {
            known.letter_counts[int(guess.at(i))-97] = -1;
        }
    }
}

/**
 * @brief Culls the word list to only contain words that are still possible solutions, based on the known information
 * 
 * @param word_list std::vector<std::string>, the list to cull
 * @param known Knowledge, the known information
 */
void cull_word_list(std::vector<std::string>& word_list, Knowledge known) {
    for (int i = word_list.size()-1; i>=0; i--) {
        bool is_valid = true;
        for (int j = 0; j<5; j++) {
            if (known.letter_counts[int(word_list.at(i).at(j))-97] < 0) {
                is_valid = false;
                break;
            } else if (known.state[j] != 0 && word_list.at(i).at(j) != known.state[j]) {
                is_valid = false;
                break;
            }
        }

        if (!is_valid) {
            word_list.erase(word_list.begin() + i);
        }
    }
}

/**
 * @brief Kernel function to get the expected information for each word in the word_list
 * 
 * @param word_list char**, the list of words to get expected information for
 * @param solution_list char**, the list of potential solutions to check each element of word_list against
 * @param n int, the number of words in the word list
 * @param k int, the number of words in the solution list
 * @param info float*, the list of expected information values (generated by this function)
 */
__global__ void get_expected_information(char *word_list, char *solution_list, int *n, int *k, float *info) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < *n) {
        char potential_guess[5] = {word_list[idx*5], word_list[idx*5+1], word_list[idx*5+2], 
                                    word_list[idx*5+3], word_list[idx*5+4]};
        int *exclusions;

        for (int i = 0; i<*k*5; i++) {
            char potential_solution[5] = {solution_list[i*5], solution_list[i*5+1], solution_list[i*5+2], 
                                        solution_list[i*5+3], solution_list[i*5+4]};

            // Find the information
            char state[5];
            char letter_counts[26] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            for (int g = 0; g<5; g++) {
                int count = 0;
                bool found = false;
                for (int s = 0; s<5; s++) {
                    if (potential_guess[g] == potential_solution[s]) { // Finding the counts gotten from the guess
                        count++;
                        found = true;
                        if (g == s) { // Update the state where necessary
                            state[g] = potential_guess[g];
                        }
                    }
                }

                // Update the letter counts if necessary
                if (letter_counts[int(potential_guess[g])-97] < count) {
                    letter_counts[int(potential_guess[g])-97] = count;
                }

                if (!found) {
                    letter_counts[int(potential_guess[g])-97] = -1;
                }
            }

            // Count excluded possible guesses
            int num_excluded = 0;
            for (int j = *n; j>=0; j--) {
                bool is_valid = true;
                for (int l = 0; l<5; l++) {
                    if (letter_counts[int(potential_guess[l]-97)] < 0) {
                        is_valid = false;
                        break;
                    } else if (state[l] != 0 && potential_guess[l] != state[l]) {
                        is_valid = false;
                        break;
                    }
                }

                if (!is_valid) {
                    num_excluded++;
                }
            }
            exclusions[i] = num_excluded;
        }

        // Average the information values and store it in the output
        float expected = 0.0;
        for (int i = 0; i<*k; i++) {
            expected += exclusions[i];
        }

        info[idx] = expected / *k;
    }
}

/**
 * @brief Makes a guess for the solution based on the amount of information that can be expected to be found by making the guess.
 * The expected information value for each 
 * 
 * @param word_list std::vector<std::string>, the list of words to guess from
 * @return std::string 
 */
std::string make_informed_guess(std::vector<std::string> word_list) {
    int temp = word_list.size();
    int *n = &temp;

    // Allocate and initialize host memory
    float *info = (float*)malloc(temp*sizeof(float));
    char *words = (char*)malloc(temp*5*sizeof(char));
    for (int i = 0; i<temp*5; i++) {
        words[i] = word_list[int(i/5)].at(i%5);
    }

    // Allocate device memory
    float *d_info;
    char *d_words, *d_sols;
    int *d_n, *d_k;
    hipMalloc(&d_info, temp*sizeof(float));
    hipMalloc(&d_words, temp*5*sizeof(char));
    hipMalloc(&d_sols, temp*5*sizeof(char));
    hipMalloc(&d_n, sizeof(int));
    hipMalloc(&d_k, sizeof(int));

    // Copy from host to device
    hipMemcpy(d_words, words, temp*5*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_sols, words, temp*5*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_n, n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_k, n, sizeof(int), hipMemcpyHostToDevice);

    // Kernel call
    get_expected_information<<<32, 512>>>(d_words, d_sols, d_n, d_k, d_info); // 32, 512
    hipDeviceSynchronize();

    // Copy data back to host
    hipMemcpy(info, d_info, temp*sizeof(float), hipMemcpyDeviceToHost);

    // Interpret data
    int max_idx = 0;
    for (int i = 0; i<temp; i++) 
        if (info[i] > info[max_idx]) 
            max_idx = i;

    // Free memory
    free(info); 
    free(words); 
    hipFree(d_info); 
    hipFree(d_words); 
    hipFree(d_n); 
    hipFree(d_k); 
    
    return word_list[max_idx];
}

/**
 * @brief Makes a guess for the solution at random
 * 
 * @param word_list std::vector<std::string>, the list of words to guess from
 * @return std::string, The randomly-made guess
 */
std::string make_random_guess(std::vector<std::string> word_list) {
    return word_list[std::rand() % word_list.size()];
}

/**
 * @brief Solves a wordle puzzle for a given solution
 * 
 * @param word std::string, the solution to solve for
 * @param path std::string, the file path for a list of words to use as the possible guesses
 * @param t char, the method to solve it with. Should be 'r' for random or 'i' to use expected information
 * @return int, the number of guesses it took to solve, or -1 if it failed
 */
int solve(std::string word, std::string path, char t, bool print) {
    bool solved = false;
    short attempts = 0;
    Knowledge known = {};
    std::vector<std::string> words = get_word_list(path, 12972); // 12972

    if (t == 'r') {
        if (print) std::cout << "Guessing '" << word << "' with random guesses..." << std::endl;
        while (attempts < 6 && !solved) {
            std::string guess = make_random_guess(words);
            update_knowledge(known, guess, word);
            int guess_idx = 0;
            for (int i = 0; i<words.size(); i++)
                if (words[i] == guess) {
                    guess_idx = i;
                    break;
                }
            words.erase(words.begin() + guess_idx);
            cull_word_list(words, known);
            if (print) {std::cout << "     "; print_guess(known, guess);}
            attempts++;
            if (guess == word)
                solved = true;
        }

        if (print) {
            std::string message = (solved)?("Solved!"):("Failed!");
            std::cout << message << std::endl;
        }
    } else if (t == 'i') {
        if (print) std::cout << "Guessing '" << word << "' with expected information..." << std::endl;
        while (attempts < 6 && !solved) {
            std::string guess = make_informed_guess(words);
            update_knowledge(known, guess, word);
            int guess_idx = 0;
            for (int i = 0; i<words.size(); i++)
                if (words[i] == guess) {
                    guess_idx = i;
                    break;
                }
            words.erase(words.begin() + guess_idx);
            cull_word_list(words, known);
            int num_remaining = words.size();
            if (print) {std::cout << "     "; print_guess(known, guess);}
            attempts++;
            if (guess == word)
                solved = true;
        }
        if (print) {
            std::string message = (solved)?("Solved!"):("Failed!");
            std::cout << message << std::endl;
        }
    } else {
        std::cout << "Invalid method type. Use 'r' for random or 'i' to use expected information." << std::endl;
    }

    return (solved)?(attempts):(-1);
}

int main(int argc, char **argv) {
    std::srand(time(0));
    printf("\n");

    // DEBUGGING
    Knowledge test_known = {};
    std::string sol = "ounce";
    std::vector<std::string> words = {"crate", "tepid", "itchy", "ounce", "store"};
    while (true) {
        std::string guess; std::cin >> guess;
        update_knowledge(test_known, guess, sol);
        cull_word_list(words, test_known);
        print_guess(test_known, guess);
        for (std::string word : words) std::cout << word << ", ";
        std::cout << std::endl;
    }
    // END DEBUGGING

    // std::vector<std::string> sols = get_word_list(argv[1], atoi(argv[2]));
    // std::vector<int> dist;
    // for (std::string sol : sols) {
    //     dist.push_back(solve(sol, argv[1], argv[3][0], (argc > 4)));
    //     if (argc > 4) std::cout << std::endl;
    // }
    // print_dist(dist);
    
    printf("\n");
    return 0;
}