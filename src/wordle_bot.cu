
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <cstdlib>
#include <string>
#include <iostream>
#include <fstream>
#include <vector>

/**
 * @brief Used to make it easier to pass the known data between functions.
 * 
 *  Knowledge.positions is a list of 5 characters, which are only initialized once the character at that 
 * position of the solution is known
 * 
 * Knowledge.letter_counts is a list of 26 shorts which represent the coutns of each letter of the alphabet that are
 * known. The count for a letter is set to -1 if it is known that that letter is not in the solution
 */
struct Knowledge {
    char positions[5];
    short letter_counts[26] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
};

/**
 * @brief Opens a list of words stored in a file (line-separated) and returns it as an array
 * 
 * @param path std::string, the path to the file to open
 * @param count int, the number of words to extract from the file (256 or the number of items in the file by default)
 * @return std::string*, the list of words extracted from the file
 */
std::vector<std::string> get_word_list(std::string path, int count=256) {
    std::vector<std::string> output;
    output.reserve(count);

    std::ifstream file(path);
    std::string word;
    int n = count;
    while (getline(file, word) && count > 0) {
        output[n-count] = word;
        count--;
    }

    return output;
}

/**
 * @brief Determines whether or not a string is a valid word for the wordle. A word is considered valid if it has five
 * characters, and each character is a letter.
 * 
 * @param word std::string, the word to validate
 * @return true if word is valid, otherwise false
 */
bool validate(std::string word) {
    bool valid = word.size() == 5;
    for (char c : word)
        if (!(int(c) >= 97 && int(c) <= 122)) //Confirms that the character is one of the ascii lowercase letters
            valid = false;
    return valid;
}

/**
 * @brief Updates the given Knowledge struct with new information including a new guess. The information will only be updated if
 * a valid guess is passed. If an invalid guess has passed, it throws an exception
 * 
 * @param info Knowledge&, the information to update
 * @param guess std::string, the new guess to get more information from
 */
void update_information(Knowledge& info, std::string guess) {
    if (!validate(guess))
        throw(guess);
    
}

/**
 * @brief Makes a guess for the solution at random
 * 
 * @return std::string, The randomly-made guess
 */
std::string make_random_guess() {

}

/**
 * @brief Makes a guess for the solution based on the amount of information that can be expected to be found by making the guess.
 * The expected information value for each 
 * 
 * @return std::string 
 */
std::string make_best_guess() {

}

int main(int argc, char **argv) {
    int test[5] = {0, 0, 0, 0, 0};
    printf("%d\n", test[2]);
    return 0;
}





