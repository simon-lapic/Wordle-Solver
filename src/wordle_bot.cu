
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <cstdlib>
#include <string>
#include <sstream>
#include <iostream>
#include <fstream>
#include <vector>

/**
 * @brief Used to make it easier to pass the known data between functions.
 * 
 *  Knowledge.positions is a list of 5 characters, which are only initialized once the character at that 
 * position of the solution is known
 * 
 * Knowledge.letter_counts is a list of 26 shorts which represent the coutns of each letter of the alphabet that are
 * known. The count for a letter is set to -1 if it is known that that letter is not in the solution
 */
struct Knowledge {
    char state[5];
    short letter_counts[26];
};

/**
 * @brief Opens a list of words stored in a file (line-separated) and returns it as an array
 * 
 * @param path std::string, the path to the file to open
 * @param count int, the number of words to extract from the file (256 or the number of items in the file by default)
 * @return std::string*, the list of words extracted from the file
 */
std::vector<std::string> get_word_list(std::string path, int count=256) {
    std::vector<std::string> output;
    output.reserve(count);

    std::ifstream file(path);
    std::string word;
    while (getline(file, word) && count > 0) {
        output.push_back(word);
        count--;
    }

    return output;
}

/**
 * @brief Determines whether or not a string is a valid word for the wordle. A word is considered valid if it has five
 * characters, and each character is a letter.
 * 
 * @param word std::string, the word to validate
 * @return true if word is valid, otherwise false
 */
bool validate(std::string word) {
    bool valid = word.size() == 5;
    for (char c : word)
        if (!(int(c) >= 97 && int(c) <= 122)) //Confirms that the character is one of the ascii lowercase letters
            valid = false;
    return valid;
}

/**
 * @brief Updates the given Knowledge struct with new information gained from a new guess. This method assumes that the 
 * solution is known to the user and the bot is running automatically. Otherwise, the Knowledge needs to be updated manually
 * 
 * @param known Knowledge&, the information to update
 * @param guess std::string, the new guess to get more information from
 * @param solution std::string, the solution being used at the moment
 */
void update_knowledge(Knowledge& known, std::string guess, std::string solution) {
    for (int i = 0; i<5; i++) {
        int count = 0;
        bool found = false;
        for (int j = 0; j<5; j++) {
            if (guess.at(i) == solution.at(j)) { // Finding the counts gotten from the guess
                count++;
                found = true;
                if (i == j) { // Update the state where necessary
                    known.state[i] = guess.at(i);
                }
            }
        }

        // Update the letter counts if necessary
        if (known.letter_counts[int(guess.at(i))-97] < count) {
            known.letter_counts[int(guess.at(i))-97] = count;
        }

        if (!found) {
            known.letter_counts[int(guess.at(i))-97] = -1;
        }
    }
}

/**
 * @brief Kernel function to get the expected information of a particular word
 * 
 * @param word_list char**, the list of words 
 */
__global__ void get_expected_information(char** word_list) {

}

/**
 * @brief Makes a guess for the solution based on the amount of information that can be expected to be found by making the guess.
 * The expected information value for each 
 * 
 * @param word_list std::vector<std::string>, the list of words to guess from
 * @return std::string 
 */
std::string make_best_guess(std::vector<std::string> word_list) {
    return "";
}

/**
 * @brief Makes a guess for the solution at random
 * 
 * @param word_list std::vector<std::string>, the list of words to guess from
 * @return std::string, The randomly-made guess
 */
std::string make_random_guess(std::vector<std::string> word_list) {
    return word_list[std::rand() % word_list.size()];
}

/**
 * @brief Prints a guess using the appropriate colors based on what information is known. A letter appears yellow if it appears in
 * the word but not at that position and green if it is at that position, otherwie gray if the letter is not in the word
 * 
 * @param known Knowledge, the knowledge known at the point the guess was made
 * @param guess std::string, the guess to print
 */
void print_guess(Knowledge known, std::string guess) {
    for (int i = 0; i<guess.size(); i++) {
        if (guess.at(i) == known.state[i]) {
            known.letter_counts[int(guess.at(i))-97]--;
        }
    }
    
    for (int i = 0; i<guess.size(); i++) {
        if (guess.at(i) == known.state[i]) {
            std::cout << "\x1B[32m" << guess.at(i) << "\033[0m"; // ANSI Green
        } else if (known.letter_counts[int(guess.at(i))-97] > 0) {
            known.letter_counts[int(guess.at(i))-97]--;
            std::cout << "\x1B[33m" << guess.at(i) << "\033[0m"; // ANSI Yellow
        } else {
            std::cout << guess.at(i);
        }
        std::cout << " ";
    }

    std::cout << std::endl;
}

int main(int argc, char **argv) {
    std::srand(time(0));
    printf("\n");
    // DEBUGGING
    std::vector<std::string> words = get_word_list("../data/wordle_words.txt", 1297200);
    Knowledge test_info = {};
    std::string sol = make_random_guess(words);
    update_knowledge(test_info, "cabal", sol);
    std::cout << sol << ": ";
    print_guess(test_info, "cabal");
    // END DEBUGGING
    printf("\n");
    return 0;
}





